#include "hip/hip_runtime.h"


#include "backproject_impl.h"


__global__ void vectorMulti(double *A, double *B, hipfftComplex *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        C[i].x = A[i] * B[i];
    }
}

void initgpu()
{
	int devCount;
	hipGetDeviceCount(&devCount);
	printf("GPU num for max %d \n",devCount);
	hipSetDevice(0);
}



double * gpusetdata_double(double *d_data,int N ,double *c_data)
{
	hipMalloc((void**) &d_data, N * sizeof(double));
	hipMemcpy(d_data, c_data, N * sizeof(double),hipMemcpyHostToDevice);
	return d_data;
}
float * gpusetdata_float(float *d_data,int N ,float *c_data)
{
	hipMalloc((void**) &d_data, N * sizeof(float));
	hipMemcpy(d_data, c_data, N * sizeof(float),hipMemcpyHostToDevice);
	return d_data;
}

void vector_Multi(double *data1, double *data2, hipfftComplex *res, int numElements)
{
    int threadsPerBlock = 512;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
	vectorMulti<<<blocksPerGrid, threadsPerBlock>>>(data1, data2, res, numElements);
}

void cpugetdata(tComplex<float> *c_outData, hipfftComplex *d_outData,int N)
{
	hipMemcpy(c_outData, d_outData, N * sizeof(hipfftComplex),hipMemcpyDeviceToHost);
}
hipfftComplex* gpumallocdata(hipfftComplex *d_outData,int N)
{
	hipMalloc((void**) &d_outData,  N * sizeof(hipfftComplex));
	return d_outData;
}



void printdatatofile(Complex *data,int N)
{
	FILE *fp= fopen("data1.out","w+");
	for(int i=0;i< 300 ;i++)
	{
		fprintf(fp,"%f %f |",data[i].real,data[i].imag);
		if(i%100==0)
			fprintf(fp,"\n");
	}
	fclose(fp);
}

