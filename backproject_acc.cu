#include "hip/hip_runtime.h"
#include "backproject_impl.h"



__device__ double tab_ftblobgetvalue(double *tabulatedValues, double val,double sampling,int xdim)
{

	int idx = (int)( ABS(val) / sampling);
	if (idx >= xdim)
		return 0.;
	else
		return tabulatedValues[idx];
}
__device__ float tab_ftblobgetvalue(float *tabulatedValues, float val,float sampling,int xdim)
{

	int idx = (int)( ABS(val) / sampling);
	if (idx >= xdim)
		return 0.;
	else
		return tabulatedValues[idx];
}
__global__ void vectorMulti(double *A, float *B, hipfftComplex *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        C[i].x = A[i] * B[i];
    }
}

__global__ void volumeMulti(float *Mconv, double *tabdata, int numElements, int xdim, double sampling , int padhdim, int pad_size, int ori_size, double padding_factor, float normftblob, int zslice)
{


    int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < numElements) {

		int k = index / zslice;
		int xyslice = index % (zslice);
		int i = xyslice / pad_size;
		int j = xyslice % pad_size;

		int kp = (k < padhdim) ? k : k - pad_size;
		int ip = (i < padhdim) ? i : i - pad_size;
		int jp = (j < padhdim) ? j : j - pad_size;
		double rval = sqrt((double) (kp * kp + ip * ip + jp * jp)) / (ori_size * padding_factor);

		Mconv[index] *= (tab_ftblobgetvalue(tabdata, rval, sampling, xdim) / normftblob);

	}
}
__global__ void volumeMulti_float(hipfftComplex *Mconv, float *tabdata, int numElements, int xdim, double sampling , int padhdim, int pad_size, int ori_size, double padding_factor, float normftblob, int zslice)
{


    int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < numElements) {

		int k = index / zslice;
		int xyslice = index % (zslice);
		int i = xyslice / pad_size;
		int j = xyslice % pad_size;

		int kp = (k < padhdim) ? k : k - pad_size;
		int ip = (i < padhdim) ? i : i - pad_size;
		int jp = (j < padhdim) ? j : j - pad_size;
		double rval = sqrt((double) (kp * kp + ip * ip + jp * jp)) / (ori_size * padding_factor);
/*
    	if (do_mask && rval > 1./(2. * padding_factor))
    		DIRECT_A3D_ELEM(Mconv, k, i, j) = 0.;*/

		Mconv[index].x *= (tab_ftblobgetvalue(tabdata, rval, sampling, xdim) / normftblob);
		Mconv[index].y =0;

	}
}
__global__ void vectorNormlize(hipfftComplex *A, long int size , long int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        A[i].x = A[i].x / size;
        A[i].y = A[i].y / size;
    }
}
__device__ float absfftcomplex(hipfftComplex A)
{
	return sqrt(A.x*A.x+A.y*A.y);
}

__global__ void fftDivide(hipfftComplex *A, double *Fnewweight, long int numElements,int xysize,int xsize,int ysize,int zsize,int xhalfsize,int max_r2)
{


    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int index2;
    if (index < numElements)
    {

    	double w;
		int k = index / xysize;
		int xyslicenum = index % (xysize);
		int i = xyslicenum / xsize;
		int j = xyslicenum % xsize;

		if (j < xhalfsize) {
			int kp, ip, jp;
			kp = (k < xhalfsize) ? k : k - zsize;
			ip = (i < xhalfsize) ? i : i - ysize;
			jp = (j < xhalfsize) ? j : j - xsize;
			index2 = j + i * xhalfsize + k * xhalfsize * ysize;
			if (kp * kp + ip * ip + jp * jp < max_r2) {

				w = XMIPP_MAX(1e-6, absfftcomplex(A[index]));
				Fnewweight[index2] = Fnewweight[index2] / w;
			}
		}

    }
}

void initgpu()
{
	int devCount;
	hipGetDeviceCount(&devCount);
	printf("GPU num for max %d \n",devCount);
}



double * gpusetdata_double(double *d_data,int N ,double *c_data)
{
	hipMalloc((void**) &d_data, N * sizeof(double));
	hipMemcpy(d_data, c_data, N * sizeof(double),hipMemcpyHostToDevice);
	return d_data;
}



float * gpusetdata_float(float *d_data,int N ,float *c_data)
{
	hipMalloc((void**) &d_data, N * sizeof(float));
	hipMemcpy(d_data, c_data, N * sizeof(float),hipMemcpyHostToDevice);
	return d_data;
}

void vector_Multi(double *data1, float *data2, hipfftComplex *res, int numElements)
{
    int threadsPerBlock = 512;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
	vectorMulti<<<blocksPerGrid, threadsPerBlock>>>(data1, data2, res, numElements);
}

void cpugetdata(tComplex<float> *c_outData, hipfftComplex *d_outData,int N)
{
	hipMemcpy(c_outData, d_outData, N * sizeof(hipfftComplex),hipMemcpyDeviceToHost);
}

void cpusenddata(hipfftComplex *d_outData, tComplex<float> *c_outData,int N)
{
	hipMemcpy(d_outData, c_outData, N * sizeof(hipfftComplex),hipMemcpyHostToDevice);
}
hipfftComplex* gpumallocdata(hipfftComplex *d_outData,int N)
{
	hipMalloc((void**) &d_outData,  N * sizeof(hipfftComplex));
	return d_outData;
}

void volume_Multi(float *data1, double *data2, int numElements, int xdim, double sampling , int padhdim, int pad_size, int ori_size, float padding_factor, double normftblob)
{
    int threadsPerBlock = 512;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    int zslice= pad_size*pad_size ;
    volumeMulti<<<blocksPerGrid, threadsPerBlock>>>(data1, data2,numElements, xdim, sampling,padhdim,pad_size,ori_size,padding_factor,normftblob,zslice);
}

void volume_Multi_float(hipfftComplex *data1, float *data2, int numElements, int xdim, double sampling , int padhdim, int pad_size, int ori_size, float padding_factor, double normftblob)
{
    int threadsPerBlock = 512;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    int zslice= pad_size*pad_size ;
    volumeMulti_float<<<blocksPerGrid, threadsPerBlock>>>(data1, data2,numElements, xdim, sampling,padhdim,pad_size,ori_size,padding_factor,normftblob,zslice);
}


void vector_Normlize(hipfftComplex *data1, long int normsize, long int numElements)
{
    int threadsPerBlock = 512;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorNormlize<<<blocksPerGrid, threadsPerBlock>>>(data1, normsize, numElements);
}

void fft_Divide(hipfftComplex *data1, double *Fnewweight, long int numElements,int xysize,int xsize,int ysize,int zsize, int halfxsize,int max_r2)
{
    int threadsPerBlock = 512;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
	fftDivide<<<blocksPerGrid, threadsPerBlock>>>(data1, Fnewweight, numElements, xysize,xsize,ysize,zsize,halfxsize, max_r2);
}


void layoutchange(hipfftComplex *data,int dimx,int dimy,int dimz, int padx, hipfftComplex *newdata)
{
	for (int z = 0; z < dimz; z++)
		for (int y = 0; y < dimy; y++) {
			memcpy(newdata + z * dimy * padx + y * padx, data + z * dimy * dimx + y * dimx, dimx * sizeof(hipfftComplex));
		}

	for(int z=0;z< dimz;z++)
	for (int y = 0; y < dimy; y++)
		for (int x = dimx; x < padx; x++) {
			int desx,desy,desz;
			if (y == 0)
				desy = 0;
			else
				desy = dimy - y;
			if(z==0)
				desz =0;
			else
				desz = dimz-z;

			desx=padx - x;

			newdata[z*padx*dimy+y * padx + x].x= newdata[desz*padx*dimy+desy * padx + desx].x;
			newdata[z*padx*dimy+y * padx + x].y= - newdata[desz*padx*dimy+desy * padx + desx].y;
		}
	/*	for(int z=0;z< NZ;z++)
		for (int x = 0; x < NX; x++)
			for (int y = 0; y < rawY; y++) {
				data2[z*NX*NY+x*NY+y].x=data[z*NX*rawY+x*rawY+y].x;
				data2[z*NX*NY+x*NY+y].y=data[z*NX*rawY+x*rawY+y].y;
			}*/
}
void layoutchangecomp(Complex *data,int dimx,int dimy,int dimz, int padx, hipfftComplex *newdata)
{

	for(int z=0;z< dimz;z++)
		for (int y = 0; y < dimy; y++)
			for (int x = 0; x < dimx; x++) {
				newdata[z*dimy*padx+y*padx+x].x=data[z*dimy*dimx+y*dimx+x].real;
				newdata[z*dimy*padx+y*padx+x].y=data[z*dimy*dimx+y*dimx+x].imag;
			}

	for(int z=0;z< dimz;z++)
	for (int y = 0; y < dimy; y++)
		for (int x = dimx; x < padx; x++) {
			int desx,desy,desz;
			if (y == 0)
				desy = 0;
			else
				desy = dimy - y;
			if(z==0)
				desz =0;
			else
				desz = dimz-z;

			desx=padx - x;

			newdata[z*padx*dimy+y * padx + x].x= newdata[desz*padx*dimy+desy * padx + desx].x;
			newdata[z*padx*dimy+y * padx + x].y= - newdata[desz*padx*dimy+desy * padx + desx].y;
		}

}
void windowFourier(hipfftComplex *d_Fconv,hipfftComplex *d_Fconv_window,int rawdim, int newdim)
{
	int winkp,winip,winjp;
	int rawkp,rawip,rawjp;
	int newdimx=newdim/2+1;
	int rawdimx=rawdim/2+1;
    for (long int k = 0, kp = 0; k<newdim; k++, kp = (k < newdimx) ? k : k - newdim) \
    	for (long int i = 0, ip = 0 ; i<newdim; i++, ip = (i < newdimx) ? i : i - newdim) \
    		for (long int j = 0, jp = 0; j<newdim; j++, jp = (j < newdimx) ? j : j - newdim)
    		{

    			winkp=(kp < 0) ? (kp + newdim) : (kp);
    			winip=(ip < 0) ? (ip + newdim) : (ip);
    			winjp = (jp < 0) ? (jp + newdim) : (jp);
    			int index1=winkp * newdimx *newdim+ winip *newdimx + winjp ;
    			rawkp=(kp < 0) ? (kp + rawdim) : (kp);
    			rawip=(ip < 0) ? (ip + rawdim) : (ip);
    			rawjp = (jp < 0) ? (jp + rawdim) : (jp);
    			int index2=rawkp * rawdimx *rawdim+ rawip *rawdimx + rawjp ;
    			d_Fconv_window[index1].x=d_Fconv[index2].x;
    			d_Fconv_window[index1].y=d_Fconv[index2].y;
    		}
}

void printdatatofile(Complex *data,int N,int dimx,int flag)
{
	FILE *fp;
	if(flag == 0)
	{
		fp= fopen("complex_gpu.out","w+");
	}
	else
	{
		fp= fopen("complex_cpu.out","w+");
	}
	for(int i=0;i< N ;i++)
	{
		//fprintf(fp,"%f %f ",data[i].real,data[i].imag);
		fprintf(fp,"%f ",data[i].real);
		if(i%dimx==0 && i!=0)
			fprintf(fp,"\n");
	}
	fclose(fp);
}
void printdatatofile(double *data,int N,int dimx,int flag)
{
	FILE *fp;
	if(flag == 0)
	{
		fp= fopen("double_gpu.out","w+");
	}
	else
	{
		fp= fopen("double_cpu.out","w+");
	}
	for(int i=0;i< N ;i++)
	{
		fprintf(fp,"%f ",data[i]);
		if(i%dimx==0 && i!=0)
			fprintf(fp,"\n");
	}
	fclose(fp);
}
void printdatatofile(float *data,int N,int dimx,int flag)
{
	FILE *fp;
	if(flag == 0)
	{
		fp= fopen("float_gpu.out","w+");
	}
	else
	{
		fp= fopen("float_cpu.out","w+");
	}
	for(int i=0;i< N ;i++)
	{
		fprintf(fp,"%f ",data[i]);
		if(i%dimx==0 && i!=0)
			fprintf(fp,"\n");
	}
	fclose(fp);
}
void printdatatofile(hipfftComplex *data,int N,int dimx,int flag)
{
	FILE *fp;
	if(flag == 0)
	{
		fp= fopen("cufftcomplex_gpu.out","w+");
	}
	else
	{
		fp= fopen("cufftcomplex_cpu.out","w+");
	}
	for(int i=0;i< N ;i++)
	{
		//fprintf(fp,"%f %f ",data[i].x,data[i].y);
		fprintf(fp,"%f ",data[i].x);
		if(i%dimx==0 && i!=0)
			fprintf(fp,"\n");
	}
	fclose(fp);
}
