#include "backproject_impl.h"



void printwholeres(hipfftComplex *out, int dimx,int dimy,int dimz) {

        printf("=====================\n");
        for (int i = 0; i < 10; i++)
                printf("%f %f \n", out[i].x, out[i].y);  //a

        for (int i = 0 + (dimx * dimy / 2); i < 10 + (dimx * dimy / 2); i++)  //b
                printf("%f %f \n", out[i].x, out[i].y);

        for (int i = dimx * dimy *(dimz/2); i < dimx * dimy *(dimz/2) + 10; i++) //c
                printf("%f %f \n", out[i].x, out[i].y);

        for (int i = dimx * dimy / 2 * dimz + (dimx * dimy / 2);
                        i < dimx * dimy / 2 * dimz + (dimx * dimy / 2) + 10; i++) //dd
                printf("%f %f \n", out[i].x, out[i].y);

        for(int i= dimx*dimy*dimz -10 ; i<dimx*dimy*dimz;i++)  // end
                printf("%f %f \n",out[i].x, out[i].y);

        for(int i= dimx*dimy*(dimz-1) ; i<dimx*dimy*(dimz-1)+10;i++)  // last slice
                printf("%f %f \n",out[i].x, out[i].y);

}

void datainit(hipfftComplex *data,int NXYZ)
{
        for (int i = 0; i < NXYZ; i++) {
                data[i].x = i % 5000 ;
                data[i].y= 0;
        }
}

void multi_plan_init(MultiGPUplan *plan, int GPU_N, size_t fullsize, int dimx,int dimy,int dimz)
{
	//MultiGPUplan plan[MAXGPU];
	for (int i = 0; i < GPU_N; i++) {
		//deviceNum[i] = i;
		plan[i].devicenum = i;
		plan[i].datasize = fullsize;
	}
	plan[0].selfoffset = 0;
	plan[1].selfoffset = dimx * dimy * (dimz / 2);
}
void multi_enable_access(MultiGPUplan *plan,int GPU_N)
{
	int can_access_peer = -100;
	hipDeviceCanAccessPeer(&can_access_peer, plan[0].devicenum,
			plan[1].devicenum);
	for (int i = 0; i < GPU_N; i++) {
		hipSetDevice(plan[i].devicenum);
		hipDeviceEnablePeerAccess((GPU_N - 1) - plan[i].devicenum, 0);
	}
	for (int i = 0; i < GPU_N; i++) {
		hipSetDevice(plan[i].devicenum);
		hipDeviceSynchronize();
	}
}
void multi_memcpy_data(MultiGPUplan *plan, hipfftComplex *f,int GPU_N,int dimx,int dimy )
{
	int offset=0;
	for (int i = 0; i < GPU_N; ++i) {
		hipSetDevice(plan[i].devicenum);
		hipMemcpyAsync(plan[i].d_Data + plan[i].selfoffset,
				f + offset,(plan[i].selfZ * dimx * dimy) * sizeof(hipfftComplex),hipMemcpyHostToDevice);
		offset += plan[0].selfZ  *dimx * dimy ;
	}
	for (int i = 0; i < GPU_N; i++) {
		hipSetDevice(plan[i].devicenum);
		hipDeviceSynchronize();
	}
}
void multi_memcpy_databack(MultiGPUplan *plan, hipfftComplex *out,int GPU_N,int dimx,int dimy)
{
	int offset = 0;
	hipSetDevice(0);
	hipMemcpyAsync(out, plan[0].d_Data,(plan[0].selfZ * dimx * dimy) * sizeof(hipfftComplex),hipMemcpyDeviceToHost);
	offset += plan[0].selfZ * dimx * dimy;
	hipMemcpyAsync(out + offset, plan[1].d_Data + offset,(plan[1].selfZ * dimx * dimy) * sizeof(hipfftComplex),hipMemcpyDeviceToHost);

	for (int i = 0; i < GPU_N; i++) {
		hipSetDevice(plan[i].devicenum);
		hipDeviceSynchronize();
	}
}

void mulit_alltoall_one(MultiGPUplan *plan, int dimx,int dimy,int dimz, int extraz,int *offsetZ)
{
	hipSetDevice(0);
	int nxy = dimx * dimy;
	int halfslice1 = (offsetZ[0]) * dimx; //Z reperesent Y
	int halfslice2 = (offsetZ[0]) * dimx * dimy;
	int sliceoffset021;
	int sliceoffset120;

	int deltanxy = 0;
	int cpysize01 = (offsetZ[1]) * dimx * sizeof(hipfftComplex);
	int cpysize10 = (offsetZ[0]) * dimx * sizeof(hipfftComplex);
	hipStream_t stream1, stream2;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);

	for (int j = 0; j < offsetZ[0]; j++) {
		sliceoffset021 = halfslice1 + deltanxy;
		sliceoffset120 = halfslice2 + deltanxy;

		hipMemcpyAsync(plan[1].d_Data + sliceoffset021, plan[0].d_Data + sliceoffset021, cpysize01,hipMemcpyDeviceToDevice, stream1);
		hipMemcpyAsync(plan[0].d_Data + sliceoffset120, plan[1].d_Data + sliceoffset120, cpysize10,hipMemcpyDeviceToDevice, stream2);

		deltanxy += nxy;
	}
	//extra for 1 z
	deltanxy = 0;

	for (int j = (dimz - extraz); j < dimz; j++) {
		hipMemcpyAsync(plan[0].d_Data + (dimz - extraz) * dimx * dimy + deltanxy,
				plan[1].d_Data + (dimz - extraz) * dimx * dimy + deltanxy,cpysize10, hipMemcpyDeviceToDevice, stream1);
		deltanxy += nxy;
	}
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);


}
void mulit_alltoall_two(MultiGPUplan *plan, int dimx,int dimy,int dimz, int extraz,int *offsetZ)
{
	int nxy = dimx * dimy;
	int halfslice1 = (offsetZ[0]) * dimx; //Z reperesent Y
	int halfslice2 = (offsetZ[0]) * dimx * dimy;
	int cpysize01 = (offsetZ[1]) * dimx * sizeof(hipfftComplex);
	int cpysize10 = (offsetZ[0]) * dimx * sizeof(hipfftComplex);
	int deltanxy = 0;
	int sliceoffset021,sliceoffset120;
	hipStream_t stream1, stream2;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);

	for (int j = 0; j < offsetZ[0]; j++) {
		sliceoffset021 = halfslice1 + deltanxy;
		sliceoffset120 = halfslice2 + deltanxy;

		hipMemcpyAsync(plan[0].d_Data + sliceoffset021,
				plan[1].d_Data + sliceoffset021, cpysize01,
				hipMemcpyDeviceToDevice, stream1);
		hipMemcpyAsync(plan[1].d_Data + sliceoffset120,
				plan[0].d_Data + sliceoffset120, cpysize10,
				hipMemcpyDeviceToDevice, stream2);
		deltanxy += nxy;
	}
	deltanxy = 0;
	for (int j = (dimz - extraz); j < dimz; j++) {
		hipMemcpyAsync(plan[1].d_Data + (dimz - extraz) * dimx * dimy + deltanxy,
				plan[0].d_Data + (dimz - extraz) * dimx * dimy + deltanxy,
				cpysize10, hipMemcpyDeviceToDevice, stream1);
		deltanxy += nxy;
	}
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);

}

void mulit_alltoall_all1to0(MultiGPUplan *plan, int dimx,int dimy,int dimz, int extraz,int *offsetZ)
{
	int nxy = dimx * dimy;
	int halfslice1 = (offsetZ[0]) * dimx; //Z reperesent Y
	int cpysize01 = (offsetZ[1]) * dimx * sizeof(hipfftComplex);
	int deltanxy = 0;
	int sliceoffset021;

	for (int j = 0; j < dimz; j++) {
		sliceoffset021 = halfslice1 + deltanxy;
		hipMemcpyAsync(plan[0].d_Data + sliceoffset021, plan[1].d_Data + sliceoffset021, cpysize01,hipMemcpyDeviceToDevice);

		deltanxy += nxy;
	}

}
void mulit_datacopy_0to1(MultiGPUplan *plan, int dimx,int dimy,int *offsetZ)
{

	int sliceoffset = (offsetZ[0]) * dimx * dimy;
	int cpysize = (offsetZ[1]) * dimx *dimy * sizeof(hipfftComplex);
	hipMemcpy(plan[1].d_Data + sliceoffset, plan[0].d_Data + sliceoffset, cpysize,hipMemcpyDeviceToDevice);

}

void multi_sync(MultiGPUplan *plan,int GPU_N)
{
	for (int i = 0; i < GPU_N; i++) {
		hipSetDevice(plan[i].devicenum);
		hipDeviceSynchronize();
	}
}

